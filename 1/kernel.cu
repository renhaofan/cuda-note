#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#define TPB 32

__device__
float distance(float x1, float x2) {
    return sqrt((x2 - x1) * (x2 - x1));
}

__global__
void distanceKernel(float *d_out, float *d_in, float ref) {
    const int i = blockIdx.x*blocckDim.x + threadIdx.x;
    const float x = d_in[i];
    d_out[i] = distance(x, ref);
    printf(" i = %2d: dist from %f to %f is %f. \n", i, ref, x, d_out[i]);
}

void distanceArray(float *out, float *in, float ref)